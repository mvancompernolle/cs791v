#include "hip/hip_runtime.h"

#include "add.h"

/*
  This is the function that each thread will execute on the GPU. The
  fact that it executes on the device is indicated by the __global__
  modifier in front of the return type of the function. After that,
  the signature of the function isn't special - in particular, the
  pointers we pass in should point to memory on the device, but this
  is not indicated by the function's signature.
 */
__global__ void add(int *a, int *b, int *c, int N) {
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_id < N) {
    c[thread_id] = a[thread_id] + b[thread_id];
  }
}

__global__ void addStriding(int *a, int *b, int *c, int N) {
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  while (thread_id < N) {
    c[thread_id] = a[thread_id] + b[thread_id];
    thread_id += blockDim.x * gridDim.x;
  }
}

