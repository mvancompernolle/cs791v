#include "hip/hip_runtime.h"
#include "BBMC.h"
#include <iostream>
#include "stdio.h"
#include <bitset>
#include <queue>
#include <boost/thread/thread.hpp>

__constant__ int numV;
__constant__ int numI;
__device__ long nodes;
__device__ int* globalMax;
__device__ unsigned int* globalC;
__device__ unsigned int* globalP;
__constant__ unsigned int* constN;
__constant__ unsigned int* constInvN;

// This is the declaration of the function that will execute on the GPU.
__device__ void printBitSet(unsigned int* bitset, int size);
__device__ int findFirstBit(unsigned int* bitset);
__device__ int getSetBitCount(unsigned int* bitset);
__device__ void intersectBitSet(unsigned int* bitset1, unsigned int* bitset2);
__device__ void flipBit(unsigned int& bitset, int pos);
__device__ void setBit(unsigned int& bitset, int pos);
__device__ void clearBit(unsigned int& bitset, int pos);
__device__ void copyBitSet(unsigned int* dest, unsigned int* src);

__global__ void maxCliqueP(int* currMax, unsigned int* N, unsigned int* invN, unsigned int* solution, unsigned int* max, unsigned int* devC,
 unsigned int* devP, unsigned int* devRecC, unsigned int* devRecP, unsigned int* devNewP, unsigned short* devU, unsigned short* devColor);
__device__ void recSearchP(unsigned int* solution, unsigned int* max, unsigned int* C, unsigned int* P,
 unsigned int* newP, unsigned short* U, unsigned short* color, int level);
__device__ void colorVertsP(unsigned int* P, unsigned short* U, unsigned short* color);
__device__ void copyBitSetP(unsigned int* dest, unsigned int* src);
__device__ unsigned int getSetBitCountP(unsigned int* bitset);
__device__ unsigned int findFirstBitP(unsigned int* bitset);
__device__ void intersectBitSetP(unsigned int* bitset1, unsigned int* bitset2);

// queue functions
__global__ void testQueue(int* queue);


void BBMC::luanchKernel(int threadId, unsigned int* hostN, unsigned int* hostInvN, unsigned int* retSol, unsigned int* retMax, int* currMax){
	std::cout << "new kernel launched: " << threadId << std::endl;

	unsigned int* devN, *devInvN, *devSolution, *devMax, *devRecP, *devRecC, *devNewP;
	unsigned short* devU, *devColor;
	thrust::device_vector<unsigned int> devC, devP;
	hipError_t err;
	hipEvent_t start, end, start2, end2;
	float elapsedTime;
	unsigned int* sol = new unsigned int[numInts * numBlocks];
	unsigned int* max = new unsigned int[numBlocks];

	if(numDevices == 1)
		hipSetDevice(1);
	else
		hipSetDevice(threadId);

	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventCreate(&start2);
	hipEventCreate(&end2);

	// get limit for stack and heap size
	err = hipDeviceSetLimit(hipLimitStackSize, 40048);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}

	err = hipMalloc( (void**) &devN, numInts * n * sizeof(unsigned int));
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devInvN, numInts * n * sizeof(unsigned int));
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devSolution, numInts * sizeof(unsigned int) * numBlocks);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devMax, sizeof(unsigned int) * numBlocks);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devRecC, sizeof(unsigned int) * numInts * n * numBlocks);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devRecP, sizeof(unsigned int) * numInts * n * numBlocks);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devNewP, sizeof(unsigned int) * numInts * n * numBlocks);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	// allocate a Mb for each block for color and U arrays
	err = hipMalloc( (void**) &devU, 548576 * numBlocks);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devColor, 548576 * numBlocks);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}

	// move the adjacency matrices to memory on the GPU
	// start timer to transfer data
	hipEventRecord( start2, 0 );

	err = hipMemcpy(devN, hostN, n * numInts * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMemcpy(devInvN, hostInvN, n * numInts * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	// move number of vertices to constant memory
	err = hipMemcpyToSymbol(HIP_SYMBOL(numV), &n, sizeof(int), 0, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(numI), &numInts, sizeof(int), 0, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(constN), &devN, sizeof(unsigned int*), 0, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(constInvN), &devInvN, sizeof(unsigned int*), 0, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}

	// allocate inital nodes for each gpu
	devC = activeC[threadId];
	devP = activeP[threadId];

	hipEventRecord( start, 0 );
	maxCliqueP<<<numBlocks, numInts>>>(currMax, devN, devInvN, devSolution, devMax, thrust::raw_pointer_cast( &devC[0] ),
	 thrust::raw_pointer_cast( &devP[0] ), devRecC, devRecP, devNewP, devU, devColor);
    hipEventRecord( end, 0 );
	hipEventSynchronize( end );

	// get solution back from kernel
  	err = hipMemcpy(sol, devSolution, numInts * numBlocks * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		std::cerr << "Error1: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
  	err = hipMemcpy(max, devMax, numBlocks * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		std::cerr << "Error2: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	// end timer that measures transfer time included
    hipEventRecord( end2, 0 );
	hipEventSynchronize( end2 );
    hipEventElapsedTime( &kernelTimes[threadId], start, end );
    std::cout << threadId << " Kernel Time (no transfer): " << kernelTimes[threadId] << std::endl;
    hipEventElapsedTime( &kernelTimesIO[threadId], start2, end2 );
    std::cout << threadId << " Kernel Time (transfer): " << kernelTimesIO[threadId] << std::endl;

	// print out maxes found in cuda for each search
	int m = 0, index = 0;
	for(int i=0; i<numBlocks; i++){
		if(max[i] > m){
			m = max[i];
			index = i;
			// std::cout << "i: " << i << " max: " << max[i] << std::endl;
		}
	}

	// printIntArray(sol + index*(numInts), 1, numInts);

	// //unsigned int* cudaSol = new unsigned int[numInts];
	// std::cout << "CUDA SOLUTION: " << threadId << std::endl;
	// for(int currInt = 0; currInt < numInts; currInt++){
	// 	// loop over each bit in the int
	// 	for(int b=0; b<sizeof(unsigned int)*8; b++){
	// 		int shift = 1 << b;
	// 		int val = sol[currInt + index*(numInts)] & shift;
	// 		if(val != 0)
	// 			std::cout << V[32 * currInt + b].index + 1 << " ";
	// 	}
	// 	std::cout << " | ";
	// }
	// std::cout << std::endl;


	// place local solution into global solution
	retMax[threadId] = max[index];
	for(int i=0; i<numInts; i++){
		retSol[(threadId * numInts) + i] = sol[index*numInts + i];
	}
}


BBMC::BBMC(int n, std::vector<std::vector<int> > A, std::vector<int> degree, int style) : MCQ(n, A, degree, style){
	// N stores the neighborhood of vertex v
	// the set of vertices that are adjacent to v
	N = new boost::dynamic_bitset<>[n];
	// invN stores the inverse neighborhood of vertex v
	// the set of vertices that are not adjacent to v
	invN = new boost::dynamic_bitset<>[n];
	V.resize(n);
}

BBMC::~BBMC(){
	if(N != NULL)
		delete[] N;
	if(invN != NULL)
		delete[] invN;
	if(kernelTimes != NULL)
		delete[] kernelTimes;
	if(kernelTimesIO != NULL)
		delete[] kernelTimesIO;
}

void BBMC::orderVertices(){
	// calculate the sum of the neighboring degrees
	for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){
			if(A[i][j] == 1)
				V[i].setNebDeg(V[i].getNebDeg() + degree[j]);
		}
	}

	// order based on style passed in
	switch(style){
		case 1:
			// order by non-increasing degree, tie-breaking on index
			std::sort(V.begin(), V.end(), Vertex::VertexCmp);
			break;
		case 2:
			// order by minimum width order
			minWidthOrder(V);
			break;
		case 3:
			// known as MCR
			// order by non-increasing degree, tie-greaking on sum of the neighborhood
			// degree nubDeg and then on index
			std::sort(V.begin(), V.end(), Vertex::MCRComparator);
			break;
	}

	int u, v;
	for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){
			u = V[i].index;
			v = V[j].index;
			N[i][j] = (A[u][v] == 1);
			invN[i][j] = (A[u][v] == 0);
		}
	}

}

void BBMC::BBColor(const boost::dynamic_bitset<>& P, int U[], int color[]){
	// copy of candidate set
	boost::dynamic_bitset<> copyP(P);
	int v;
	int colorClass = 0;
	int i = 0;

	while(copyP.count() != 0){
		colorClass++;
		boost::dynamic_bitset<> Q(copyP);

		while(Q.count() != 0){
			// return the index of the first set bit
			v = Q.find_first();
			
			// remove v from Q and copyP
			copyP[v] = 0;
			Q[v] = 0;

			// perform a bitwise and operation
			// Q becomes set of vertices that are in Q but not adjacent to v
			Q &= invN[v];
			U[i] = v;
			color[i++] = colorClass;
		}
	}
}

void BBMC::printBitSet(const boost::dynamic_bitset<>& bitset) const{
	for(int i=0; i<bitset.size(); i++){
		std::cout << bitset[i] << " ";
		if(i%32 == 0 && i != 0)
			std::cout << " | ";
	}
	std::cout << std::endl;
}

void BBMC::printIntArray(unsigned int* arr, int n, int numInts) const{
	// loop over each row
	for(int i=0; i<n; i++){
		// loop over each int
		for(int currInt = 0; currInt < numInts; currInt++){
			// loop over each bit in the int
			for(int b=0; b<sizeof(unsigned int)*8; b++){
				int shift = 1 << b;
				int val = arr[(i*numInts) + currInt] & shift;
				if(val != 0)
					val = 1;
				std::cout << val << " ";
			}
			std::cout << " | ";
		}
		std::cout << std::endl;
	}
}

////////////////////// CUDA FUNCTIONS ///////////////////////////////////////////////////////////

__device__ int findFirstBit(unsigned int* bitset){
	int pos = 0, index = 0;;
	for(int i=0; i < numI; i++){
		pos = __ffs(bitset[i]);
		//printf("pos %d\n", pos);
		if(pos != 0){
			index = i;
			break;
		}
	}
	return pos + (index * 32) - 1;
}

__device__ int getSetBitCount(unsigned int* bitset){
	int count = 0;
	for(int i=0; i<numI; i++){
		count += __popc(bitset[i]);
	}
	return count;
}

__device__ void intersectBitSet(unsigned int* bitset1, unsigned int* bitset2){
	for(int i=0; i<numI; i++){
		bitset1[i] &= bitset2[i];
	}
}

__device__ void flipBit(unsigned int& bitset, int pos){
	bitset ^= (1u << pos);
}

__device__ void setBit(unsigned int& bitset, int pos){
	bitset |= (1u << pos);
}

__device__ void clearBit(unsigned int& bitset, int pos){
	bitset &= ~(1u << pos);
}

__device__ void copyBitSet(unsigned int* dest, unsigned int* src){
	for(int i=0; i<numI; i++){
		dest[i] = src[i];
	}
}

__device__ void printBitSet(unsigned int* bitset, int size){

	// loop over each int
	for(int currInt = 0; currInt < size; currInt++){
		// loop over each bit in the int
		for(int b=0; b<sizeof(unsigned int)*8; b++){
			int shift = 1 << b;
			int val = bitset[currInt] & shift;
			if(val != 0)
				val = 1;
			printf("%u ", val);
		}
		printf(" | ");
	}
}

//////////////////////////// CUDA PARALLEL FUNCITONS /////////////////////////
void BBMC::searchParallel(int num){
	hipError_t err;
	numInts = (n+sizeof(int)*8-1)/(sizeof(int)*8);
	// get the number of devices
	numDevices = num;
	if(num != 1){
		err = hipGetDeviceCount(&numDevices);
		if (err != hipSuccess) {
			std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
			exit(1);
		}
	}
	std::cout << "Num Devices: " << numDevices << std::endl;
	numBlocks = n;
	activeC.resize(numDevices);
	activeP.resize(numDevices);

	// allocate memory for timing results
	kernelTimes = new float[numDevices];
	kernelTimesIO = new float[numDevices];

	timeval tod1, tod2;
	gettimeofday(&tod1, NULL);

	// calculate the number of ints needed per vertex on gpu
	int r = n % (sizeof(int)*8);
	std::cout << numInts << " " << r << " " << n << " " << (sizeof(unsigned int)*8) << std::endl;
	unsigned int* hostN = new unsigned int[numInts*n];
	unsigned int* hostInvN = new unsigned int[numInts*n];
	unsigned int* sol = new unsigned int[numDevices * numInts];
	unsigned int* max = new unsigned int[numDevices];

	for(int i=0; i<n; i++){
		N[i].resize(n);
		invN[i].resize(n);
		V[i].index = i;
		V[i].degree = degree[i];

	}

	// order vertices
	orderVertices();

	// default all values to 0
	for(int r=0; r<n; r++){
		for(int i=0; i<numInts; i++){
			hostN[r*numInts + i] = 0;
			hostInvN[r*numInts + i] = 0;
		}
	}

	// copy the adjacency matrices into 1D arrays
	for(int v=0; v<n; v++){
		for(int i=0; i<numInts; i++){
			for(int j=0; j<sizeof(unsigned int)*8; j++){

				if(r != 0 && i == numInts-1 && j == r)
					break;

				// assign each bit the the integer array
				hostN[(v*numInts) + i] |= (N[v][i * 32 + j] << j);

				int invVal = 0;
				if(N[v][i * 32 + j] == 0)
					invVal = 1;
				hostInvN[(v*numInts) + i] |= (invVal << j);
			}
		}
	}

	generateInitialNodes(numBlocks, numDevices);

	size_t stackLimit, heapLimit;
	err = hipDeviceGetLimit(&stackLimit, hipLimitStackSize);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipDeviceGetLimit(&heapLimit, hipLimitMallocHeapSize);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	std::cout << "Stack Limit: " << stackLimit << " Heap Limit: " << heapLimit << std::endl;

	// allocate unified memory to share current max size
	int* currentMax;
	hipMallocManaged(&currentMax, sizeof(int));
	*currentMax = 1;

	// create threads to launch a kernel on each gpu
	boost::thread threads[numDevices];

	// size for bitwise operations
	for(int i=0; i<numDevices; i++){
		threads[i] = boost::thread(&BBMC::luanchKernel, this, i, hostN, hostInvN, sol, max, currentMax);
	}
	gettimeofday(&tod2, NULL);
	preProcessing = todiff(&tod2, &tod1)/1000;

	// join threads
	for(int i=0; i<numDevices; i++){
		threads[i].join();
	} 

	// print out maxes found in cuda for each search
	int m = 0, index = 0;
	for(int i=0; i<numDevices; i++){
		if(max[i] > m){
			m = max[i];
			index = i;
		}
	}
	// std::cout << "MAX SIZE: " << *currentMax << std::endl;
	maxSize = *currentMax;
	// printIntArray(sol + index*(numInts), 1, numInts);

	unsigned int* cudaSol = new unsigned int[numInts];
	std::cout << "CUDA SOLUTION FINAL: " << std::endl;
	for(int currInt = 0; currInt < numInts; currInt++){
		// loop over each bit in the int
		for(int b=0; b<sizeof(unsigned int)*8; b++){
			int shift = 1 << b;
			int val = sol[currInt + index*(numInts)] & shift;
			if(val != 0)
				std::cout << V[32 * currInt + b].index + 1 << " ";
		}
		std::cout << " | ";
	}
	std::cout << std::endl;

	// clean up
	delete[] hostN;
	delete[] hostInvN;
	delete[] sol;
	delete[] max;
}

void BBMC::generateInitialNodes(int numBlocks, int numDevices){
	boost::dynamic_bitset<> C(n);
	boost::dynamic_bitset<> P(n);
	boost::dynamic_bitset<> newP;
	std::queue<boost::dynamic_bitset<>> activeCBitset;
	std::queue<boost::dynamic_bitset<>> activePBitset;
	int m, v;
	unsigned int c = 0, p = 0;
	int num = numBlocks * numDevices;

	// initialize starting node
	for(int i=0; i<C.size(); i++){
		C[i] = 0;
		P[i] = 1;
	}

	// generate initial branches for the graph (equal to number of vertices)
	m = P.count();
	int U[m];
	int color[m];
	BBColor(P, U, color);

	// iterate over the candidate set
	for(int i=m-1; i>= 0 && activeCBitset.size()<num-1; i--){

		// select a vertex from P and add it to the current clique
		// newP is set of vertices in the candidate set P that are adjacent to v
		newP = P;
		int v = U[i];
		C[v] = 1;

		// perform bitwise and (fast for set of element that reside in word boundaries)
		newP &= N[v];

		activeCBitset.push(C);
		activePBitset.push(newP);

		// remove v from P and C when returning
		C[v] = 0;
		P[v] = 0;
	}
	if(P.count() > 0){
		activeCBitset.push(C);
		activePBitset.push(P);
	}

	// expand nodes in queue to enqueue more sub branches and increase parallelism
	while(activeCBitset.size() < num){
		// get first nodes in queue
		C = activeCBitset.front();
		activeCBitset.pop();
		P = activePBitset.front();
		activePBitset.pop();

		// color the nodes for optimization
		m = P.count();
		BBColor(P, U, color);

		// iterate over part of the candidate set (lower i's usually get bounded out)
		for(int i=m-1; i>= m/2 && activeCBitset.size()<num-1; i--){

			// select a vertex from P and add it to the current clique
			// newP is set of vertices in the candidate set P that are adjacent to v
			newP = P;
			int v = U[i];
			C[v] = 1;

			// perform bitwise and (fast for set of element that reside in word boundaries)
			newP &= N[v];

			// add node to the queue for GPU
			activeCBitset.push(C);
			activePBitset.push(newP);

			// remove v from P and C when returning
			C[v] = 0;
			P[v] = 0;
		}
		// push the node that was initially popped off back on
		activeCBitset.push(C);
		activePBitset.push(P);
	}

	std::cout << std::endl << "num blocks: " << activeCBitset.size() << std::endl;

	// convert bitsets to GPU format

	int count = 0;
	int limit = numBlocks;
	int index = 0;
	while(activeCBitset.size() > 0){
		C = activeCBitset.front();
		newP = activePBitset.front();
		activeCBitset.pop();
		activePBitset.pop();
		for(int i=0; i<numInts; i++){
			c = 0;
			p = 0;
			for(int j=0; j<sizeof(unsigned int)*8; j++){
				//std::cout << "j " << j << " i " << i << std::endl;
				// assign each bit the the integer array
				c |= C[i*32 + j] << j;
				p |= newP[i*32 + j] << j;
				for(int b=0; b<32; b++){
					int num = (c & (1 << b));
					if(num != 0)
						num = 1;
					//std::cout << num << " ";
				}
				//std::cout << std::endl;
			}		
			// std::cout << "c " << c << " p " <<  p << std::endl;
			activeC[index%numDevices].push_back(c);
			activeP[index%numDevices].push_back(p);
		}
		// std::cout << count << " " << limit << " " << index << std::endl;
		count++;
		if(count == limit){
			index++;
			count = 0;
		}
	}


	// while(activeCBitset.size() > 0){
	// 	C = activeCBitset.front();
	// 	activeCBitset.pop();
	// 	P = activePBitset.front();
	// 	activePBitset.pop();
	// 	std::cout << "node C: " << std::endl;
	// 	printBitSet(C);
	// 	std::cout << "node P: " << std::endl;
	// 	printBitSet(P);
	// }
}

__global__ void maxCliqueP(int* currMax, unsigned int* devN, unsigned int* devInvN, unsigned int* solution, unsigned int* max, unsigned int* devC,
 unsigned int* devP, unsigned int* devRecC, unsigned int* devRecP, unsigned int* newP, unsigned short* devU,
 unsigned short* devColor) {

	//time1 = clock();
	__shared__ unsigned int* C;
	__shared__ unsigned int* P;
	__shared__ unsigned short* U;
	__shared__ unsigned short* color;
	//numI = 1;
	//printf("numI: %u\n", numI);

	// have each block 
	if(threadIdx.x == 0){
		nodes = 0;
		max[blockIdx.x] = 0;
		C = devC + (numI * blockIdx.x);
		P = devP + (numI * blockIdx.x);
		U = devU + (blockIdx.x * 548576/sizeof(unsigned short));
		color = devColor + (blockIdx.x * 548576/sizeof(unsigned short));
		globalMax = currMax;
	}
	__syncthreads();

	copyBitSetP(devRecC + (blockIdx.x * numV * numI), C);
	copyBitSetP(devRecP + (blockIdx.x * numV * numI), P);
	copyBitSetP(newP + (blockIdx.x * numV * numI), P);

	__syncthreads();

	recSearchP(solution, max, devRecC + (blockIdx.x * numV * numI) + numI, 
		devRecP + (blockIdx.x * numV * numI) + numI, newP + (blockIdx.x * numV * numI), U, color, 0);
}

__device__ void recSearchP(
 unsigned int* solution, unsigned int* max, unsigned int* C, unsigned int* P, unsigned int* newP,
 unsigned short* U, unsigned short* color, int level){

	int newPNum, cNum;
	int m, currSize, v;

	// copy C and P
	copyBitSetP(C, C-numI);
	copyBitSetP(P, newP);

 	__syncthreads();
	// ahve only a single thread incr the num nodes
	if(threadIdx.x == 0 && blockIdx.x == 0){
		nodes++;
	}

	newP += numI;

	m = getSetBitCountP(P);
	currSize = getSetBitCountP(C);

	colorVertsP(P, U, color);

	__syncthreads();

	// iterate over the candidate set
	for(int i=m-1; i>=0; i--){

		if(color[i] + currSize <= *globalMax){
			return;
		}

		// copy the candidate set
		copyBitSetP(newP, P);

		// pick a candidate
		v = U[i];
		if(threadIdx.x == 0){
			setBit(C[v/32], v%32);
		}

		// create the new candidate set
		intersectBitSetP(newP, constN + (v * numI));

		__syncthreads();

		// get the set bits for the candidate set and the current set
		newPNum = getSetBitCountP(newP);

		currSize++;

		// if maximal, check for maximum
		if(newPNum == 0 && currSize > *globalMax){

			if(threadIdx.x == 0){
				// save the new max size so that it is shared among blocks
				atomicMax(globalMax, currSize);
				max[blockIdx.x] = currSize;
			}
			copyBitSetP(solution + (blockIdx.x*numI), C);
		}
		else if(newPNum > 0){
			recSearchP(solution, max, C + numI, P + numI, newP, U + m, color + m, level + 1);
		}

		__syncthreads();

		// remove v from P and C
		if(threadIdx.x == 0){
			clearBit(C[v/32], v%32);
			clearBit(P[v/32], v%32);
		}	
		currSize--;
		__syncthreads();
		// return;
	}
}

__device__ void colorVertsP(unsigned int* P, unsigned short* U, unsigned short* color){

	// copy the candidate set
	//printf("thread: %d\n", threadIdx.x);
	__shared__ int colorClass;
	__shared__ int i;
	__shared__ unsigned int copyP[47];
	__shared__ unsigned int Q[47];
	int v;

	copyBitSetP(copyP, P);

	// have main thread init values to zero
	if(threadIdx.x == 0){
		colorClass = 0;
		i = 0;
	}

	__syncthreads();

	while(getSetBitCountP(copyP) != 0){

		if(threadIdx.x == 0){
			colorClass++;
		}

		// copy the candidate set
		copyBitSetP(Q, copyP);

		__syncthreads();

		while(getSetBitCountP(Q) != 0){

			__syncthreads();

			// return the index of the first set bit
			v = findFirstBitP(Q);

			__syncthreads();

			// remove v from Q and copyP
			if(threadIdx.x == 0){
				clearBit(copyP[v/32], v%32);
				clearBit(Q[v/32], v%32);
			}

			__syncthreads();

			intersectBitSetP(Q, &constInvN[(v) * numI]);

			__syncthreads();

			if(threadIdx.x == 0){
				U[i] = v;
				color[i++] = colorClass;
			}

			__syncthreads();

		}
		__syncthreads();
	}
}

__device__ void copyBitSetP(unsigned int* dest, unsigned int* src){
	dest[threadIdx.x] = src[threadIdx.x];
}

__device__ unsigned int getSetBitCountP(unsigned int* bitset){

	__shared__ unsigned int work[47];

	work[threadIdx.x] = __popc(bitset[threadIdx.x]);

	int size = numI;
	for(int s=numI/2; s>0; s>>=1){
		__syncthreads();

	    if(threadIdx.x < s)
	      work[threadIdx.x] += work[threadIdx.x+s];

		// have the first thread do one additional add if the current size is odd
		if(size&0x0001 == 0x0001 && threadIdx.x == 0)
			work[threadIdx.x] += work[size-1];

		size /= 2;
	}

	__syncthreads();

	return work[0];
}

__device__ unsigned int findFirstBitP(unsigned int* bitset){

	__shared__ unsigned int first;
	first = 10000;

	__syncthreads();

	// have each thread get bit pos in int
	unsigned int pos = __ffs(bitset[threadIdx.x]);

	// set atomic min if bit was found
	if(pos != 0){
		// calculate overall position in bitstring and attempt to set to min
		pos += (threadIdx.x * 32) - 1;
		if(pos < first)
			atomicMin(&first, pos);
	}

	__syncthreads();

	return first;
}

__device__ void intersectBitSetP(unsigned int* bitset1, unsigned int* bitset2){
	bitset1[threadIdx.x] &= bitset2[threadIdx.x];
}

////////////////////////////// QUEUE FUNCTIONS /////////////////////////////
void BBMC::queueFcn(){
	int* devQueue;

	hipError_t err = hipMalloc( (void**) &devQueue, 1024 * sizeof(int));
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}

	testQueue<<<1, 32>>>(devQueue);
	hipDeviceSynchronize();
}

__device__ int qMaxSize = 1024;
__device__ unsigned int pos;
__global__ void testQueue(int* queue){

	// __shared__ unsigned int arr[35];
	// __shared__ unsigned int work[35];
	// arr[10] = 1 << 11;
	// arr[5] = 1 << 5;
	// if(threadIdx.x == 0){
	// 	printBitSet(arr, 35);
	// 	printf("\n\n");
	// }

	// __syncthreads();

	// int first = findFirstBitPP(arr);

	// __syncthreads();

	// if(threadIdx.x == 0){
	// 	printf("bit Pos: %d\n", first);
	// }

	// __syncthreads();

	// int num = getSetBitCountPP(arr);

	// __syncthreads();

	// if(threadIdx.x == 0){
	// 	printf("bit count: %d\n", num);
	// }

	// printf("queue\n");

	// pos = 0;

	// for(int i=0; i<100; i++){
	// 	printf("enqueueing: %d\n", atomicAdd(&pos, 1));
	// }

	// for(int i=0; i<150; i++){
	// 	printf("dequeueing: %d\n", atomicSub(&pos, 1));
	// }

}

		// unsigned int c = 0, p = 0;
		// for(int i=0; i<numInts; i++){
		// 	for(int j=0; j<sizeof(unsigned int)*8; j++){
		// 		//std::cout << "j " << j << " i " << i << std::endl;
		// 		// assign each bit the the integer array
		// 		c |= C[i*32 + j] << j;
		// 		p |= newP[i*32 + j] << j;
		// 		for(int b=0; b<32; b++){
		// 			int num = (c & (1 << b));
		// 			if(num != 0)
		// 				num = 1;
		// 			//std::cout << num << " ";
		// 		}
		// 		//std::cout << std::endl;
		// 	}		
		// 	//std::cout << "c " << c << " p " <<  p << std::endl;
		// 	activeC.push_back(c);
		// 	activeP.push_back(p);
		// 	c = 0;
		// 	p = 0;
		// }