#include "hip/hip_runtime.h"
#include "BBMC.h"
#include <iostream>
#include "stdio.h"
#include <bitset>

__constant__ int numV;
__constant__ int numI;
// Limit of about 700 vertices
__device__ __constant__ unsigned int constInvN[16000];
//__device__ float time1, time2;
__device__ long nodes;
__device__ int maxSize;
__device__ unsigned int* globalC;
__device__ unsigned int* globalP;
__device__ unsigned int* globalN;

// This is the declaration of the function that will execute on the GPU.
__device__ void printBitSet(unsigned int* bitset, int size);
__device__ int findFirstBit(unsigned int* bitset);
__device__ int getSetBitCount(unsigned int* bitset);
__device__ void intersectBitSet(unsigned int* bitset1, unsigned int* bitset2);
__device__ void flipBit(unsigned int& bitset, int pos);
__device__ void setBit(unsigned int& bitset, int pos);
__device__ void clearBit(unsigned int& bitset, int pos);
__device__ void copyBitSet(unsigned int* dest, unsigned int* src);

__global__ void maxCliqueP(unsigned int* N, unsigned int* solution,
 unsigned int* max, unsigned int* devC, unsigned int* devP, unsigned int* devRecC, unsigned int* devRecP, unsigned int* devNewP);
__device__ void recSearchP(unsigned int* N, unsigned int* solution, unsigned int* max, unsigned int* C, unsigned int* P, unsigned int* newP);
__device__ void colorVertsP(unsigned int* P, unsigned int* U, unsigned int* color);
__device__ void copyBitSetP(unsigned int* dest, unsigned int* src);


/*
Algorithm description:
- Vertices are selected from the candidate set to add to the current clique in non-decreasing color order
	with a color cut-off.
- Bitset encoding of MCSa with the following features:
	1. The "BB" in "BB-MaxClique is for "Bit Board". Sets are represented using bit string.
	2. BBMC color sthe candidate set using a static sequential ordering, the ordering set at the top of search
	3. BBMC represents the neighborhood of a vertex and its inverse neighborhood as bit strings, rather 
		than using an adjacency matrix and its complement.
	4. When coloring takes place a color class perspective is taken, determining what vertices can be placed 
		in a color class together, before moving onto the next color class. 
*/

BBMC::BBMC(int n, std::vector<std::vector<int> > A, std::vector<int> degree, int style) : MCQ(n, A, degree, style){
	// N stores the neighborhood of vertex v
	// the set of vertices that are adjacent to v
	N = new boost::dynamic_bitset<>[n];
	// invN stores the inverse neighborhood of vertex v
	// the set of vertices that are not adjacent to v
	invN = new boost::dynamic_bitset<>[n];
	V.resize(n);
	numInts = 0;
}

BBMC::~BBMC(){
	if(N != NULL)
		delete[] N;
	if(invN != NULL)
		delete[] invN;
}

void BBMC::orderVertices(){
	// calculate the sum of the neighboring degrees
	for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){
			if(A[i][j] == 1)
				V[i].setNebDeg(V[i].getNebDeg() + degree[j]);
		}
	}

	// order based on style passed in
	switch(style){
		case 1:
			// order by non-increasing degree, tie-breaking on index
			std::sort(V.begin(), V.end(), Vertex::VertexCmp);
			break;
		case 2:
			// order by minimum width order
			minWidthOrder(V);
			break;
		case 3:
			// known as MCR
			// order by non-increasing degree, tie-greaking on sum of the neighborhood
			// degree nubDeg and then on index
			std::sort(V.begin(), V.end(), Vertex::MCRComparator);
			break;
	}

	int u, v;
	for(int i=0; i<n; i++){
		for(int j=0; j<n; j++){
			u = V[i].index;
			v = V[j].index;
			N[i][j] = (A[u][v] == 1);
			invN[i][j] = (A[u][v] == 0);
		}
	}

}

void BBMC::BBColor(const boost::dynamic_bitset<>& P, int U[], int color[]){
	// copy of candidate set
	boost::dynamic_bitset<> copyP(P);
	int v;
	int colorClass = 0;
	int i = 0;

	while(copyP.count() != 0){
		colorClass++;
		boost::dynamic_bitset<> Q(copyP);

		while(Q.count() != 0){
			// return the index of the first set bit
			v = Q.find_first();
			
			// remove v from Q and copyP
			copyP[v] = 0;
			Q[v] = 0;

			// perform a bitwise and operation
			// Q becomes set of vertices that are in Q but not adjacent to v
			Q &= invN[v];
			U[i] = v;
			color[i++] = colorClass;
		}
	}
}

void BBMC::printBitSet(const boost::dynamic_bitset<>& bitset) const{
	for(int i=0; i<bitset.size(); i++){
		std::cout << bitset[i] << " ";
		if(i%32 == 0 && i != 0)
			std::cout << " | ";
	}
	std::cout << std::endl;
}

void BBMC::printIntArray(unsigned int* arr, int n, int numInts) const{
	// loop over each row
	for(int i=0; i<n; i++){
		// loop over each int
		for(int currInt = 0; currInt < numInts; currInt++){
			// loop over each bit in the int
			for(int b=0; b<sizeof(unsigned int)*8; b++){
				int shift = 1 << b;
				int val = arr[(i*numInts) + currInt] & shift;
				if(val != 0)
					val = 1;
				std::cout << val << " ";
			}
			std::cout << " | ";
		}
		std::cout << std::endl;
	}
}

////////////////////// CUDA FUNCTIONS ///////////////////////////////////////////////////////////

__device__ int findFirstBit(unsigned int* bitset){
	int pos = 0, index = 0;;
	for(int i=0; i < numI; i++){
		pos = __ffs(bitset[i]);
		//printf("pos %d\n", pos);
		if(pos != 0){
			index = i;
			break;
		}
	}
	return pos + (index * 32) - 1;
}

__device__ int getSetBitCount(unsigned int* bitset){
	int count = 0;
	for(int i=0; i<numI; i++){
		count += __popc(bitset[i]);
	}
	return count;
}

__device__ void intersectBitSet(unsigned int* bitset1, unsigned int* bitset2){
	for(int i=0; i<numI; i++){
		bitset1[i] &= bitset2[i];
	}
}

__device__ void flipBit(unsigned int& bitset, int pos){
	bitset ^= (1u << pos);
}

__device__ void setBit(unsigned int& bitset, int pos){
	bitset |= (1u << pos);
}

__device__ void clearBit(unsigned int& bitset, int pos){
	bitset &= ~(1u << pos);
}

__device__ void copyBitSet(unsigned int* dest, unsigned int* src){
	for(int i=0; i<numI; i++){
		dest[i] = src[i];
	}
}

__device__ void printBitSet(unsigned int* bitset, int size){

	// loop over each int
	for(int currInt = 0; currInt < size; currInt++){
		// loop over each bit in the int
		for(int b=0; b<sizeof(unsigned int)*8; b++){
			int shift = 1 << b;
			int val = bitset[currInt] & shift;
			if(val != 0)
				val = 1;
			printf("%u ", val);
		}
		printf(" | ");
	}
}

//////////////////////////// CUDA PARALLEL FUNCITONS /////////////////////////
void BBMC::searchParallel(){
	hipError_t err;

	for(int i=0; i<n; i++){
		N[i].resize(n);
		invN[i].resize(n);
		V[i].index = i;
		V[i].degree = degree[i];

	}

	// order vertices
	orderVertices();

/*
std::cout << "N: " << std::endl;
for(int i=0; i<n; i++)
	printBitSet(N[i]);
std::cout << "invN: " << std::endl;
for(int i=0; i<n; i++)
	printBitSet(invN[i]);
std::cout << std::endl;
std::cout << "V: (index / degree) " << std::endl;
for(Vertex& v: V){
	std::cout << v.index << "-" << v.degree << " ";
}
std::cout << std::endl;
*/
	// calculate the number of ints needed per vertex on gpu
	numInts = (n+sizeof(int)*8-1)/(sizeof(int)*8);
	int r = n % (sizeof(int)*8);
	std::cout << numInts << " " << r << " " << n << " " << (sizeof(unsigned int)*8) << std::endl;
	unsigned int* hostN = new unsigned int[numInts*n];
	unsigned int* hostInvN = new unsigned int[numInts*n];
	unsigned int* sol = new unsigned int[numInts * n];
	unsigned int* max = new unsigned int[n];
	unsigned int* devN, *devInvN, *devSolution, *devMax;

	// need to preallocate memory for recursive calls
	unsigned int* devRecC, *devRecP, *devNewP;

	hipSetDevice(1);

	// get limit for stack and heap size
	err = hipDeviceSetLimit(hipLimitStackSize, 50048);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipDeviceSetLimit(hipLimitMallocHeapSize, 1684354560);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	size_t stackLimit, heapLimit;
	err = hipDeviceGetLimit(&stackLimit, hipLimitStackSize);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipDeviceGetLimit(&heapLimit, hipLimitMallocHeapSize);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	std::cout << "Stack Limit: " << stackLimit << " Heap Limit: " << heapLimit << std::endl;

	err = hipMalloc( (void**) &devN, numInts * n * sizeof(unsigned int));
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devSolution, numInts * sizeof(unsigned int) * n);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devMax, sizeof(unsigned int) * n);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devRecC, sizeof(unsigned int) * numInts * n * n);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devRecP, sizeof(unsigned int) * numInts * n * n);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMalloc( (void**) &devNewP, sizeof(unsigned int) * numInts * n * n);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}

	// default all values to 0
	for(int r=0; r<n; r++){
		for(int i=0; i<numInts; i++){
			hostN[r*numInts + i] = 0;
			hostInvN[r*numInts + i] = 0;
		}
	}

	// copy the adjacency matrices into 1D arrays
	for(int v=0; v<n; v++){
		for(int i=0; i<numInts; i++){
			for(int j=0; j<sizeof(unsigned int)*8; j++){

				if(i == numInts-1 && j == r)
					break;

				// assign each bit the the integer array
				hostN[(v*numInts) + i] |= (N[v][i * 32 + j] << j);

				int invVal = 0;
				if(N[v][i * 32 + j] == 0)
					invVal = 1;
				hostInvN[(v*numInts) + i] |= (invVal << j);
			}
		}
	}

	// move the adjacency matrices to constant memory on the GPU
	err = hipMemcpy(devN, hostN, n * numInts * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(constInvN), hostInvN, n * numInts * sizeof(unsigned int));
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}

	// move number of vertices to constant memory
	err = hipMemcpyToSymbol(HIP_SYMBOL(numV), &n, sizeof(int), 0, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(numI), &numInts, sizeof(int), 0, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}

	generateInitialNodes();

	// allocated nodes on the gpu
	thrust::device_vector<unsigned int> devC = activeC;
	thrust::device_vector<unsigned int> devP = activeP;
	std::cout << "sizes: " << activeC.size() << " " << activeP.size() << std::endl;

	//printIntArray(hostN, n, numInts);
	//std::cout << std::endl;
	//printIntArray(hostInvN, n, numInts);
	hipEvent_t start, end;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord( start, 0 );
	maxCliqueP<<<n, numInts>>>(devN, devSolution, devMax, 
		thrust::raw_pointer_cast( &devC[0] ), thrust::raw_pointer_cast( &devP[0] ), devRecC, devRecP, devNewP);
    hipEventRecord( end, 0 );
    hipEventSynchronize( end );
    hipEventElapsedTime( &elapsedTime, start, end );
    std::cout << "Kernel Time: " << elapsedTime << std::endl;

	// get solution back from kernel
  	err = hipMemcpy(sol, devSolution, numInts * n * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}
  	err = hipMemcpy(max, devMax, n * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}

	// print out maxes found in cuda for each search
	int m = 0, index = 0;
	for(int i=0; i<n; i++){
		if(max[i] > m){
			m = max[i];
			index = i;
			std::cout << "i: " << i << " max: " << max[i] << std::endl;
		}
	}
	printIntArray(sol + index*(numInts), 1, numInts);

	//unsigned int* cudaSol = new unsigned int[numInts];
	std::cout << "CUDA SOLUTION: " << std::endl;
	for(int currInt = 0; currInt < numInts; currInt++){
		// loop over each bit in the int
		for(int b=0; b<sizeof(unsigned int)*8; b++){
			int shift = 1 << b;
			int val = sol[currInt + index*(numInts)] & shift;
			if(val != 0)
				std::cout << V[32 * currInt + b].index + 1 << " ";
		}
		std::cout << " | ";
	}
	std::cout << std::endl;
}

void BBMC::generateInitialNodes(){
	boost::dynamic_bitset<> C(n);
	boost::dynamic_bitset<> P(n);
	boost::dynamic_bitset<> newP;

	for(int i=0; i<C.size(); i++){
		C[i] = 0;
		P[i] = 1;
	}

	int m = P.count();
	int U[m];
	int color[m];
	BBColor(P, U, color);

	// iterate over the candidate set
	for(int i=n-1; i>= 0; i--){

		// select a vertex from P and add it to the current clique
		// newP is set of vertices in the candidate set P that are adjacent to v
		newP = P;
		int v = U[i];
		C[v] = 1;

		// perform bitwise and (fast for set of element that reside in word boundaries)
		newP &= N[v];

		//boost::to_block_range(C, std::back_inserter(vecC));		
		//boost::to_block_range(newP, std::back_inserter(vecP));

				//hostN[(v*numInts) + i] |= (N[v][i * 32 + j] << j);

		//std::cout << "node C: " << std::endl;
		//printBitSet(C);
		//std::cout << "node P: " << std::endl;
		//printBitSet(newP);	

		unsigned int c = 0, p = 0;
		for(int i=0; i<numInts; i++){
			for(int j=0; j<sizeof(unsigned int)*8; j++){
				//std::cout << "j " << j << " i " << i << std::endl;
				// assign each bit the the integer array
				c |= C[i*32 + j] << j;
				p |= newP[i*32 + j] << j;
				for(int b=0; b<32; b++){
					int num = (c & (1 << b));
					if(num != 0)
						num = 1;
					//std::cout << num << " ";
				}
				//std::cout << std::endl;
			}		
			//std::cout << "c " << c << " p " <<  p << std::endl;
			activeC.push_back(c);
			activeP.push_back(p);
			c = 0;
			p = 0;
		}

		// remove v from P and C when returning
		C[v] = 0;
		P[v] = 0;
	}
}

__global__ void maxCliqueP(unsigned int* N, unsigned int* solution, unsigned int* max,
 unsigned int* devC, unsigned int* devP, unsigned int* devRecC, unsigned int* devRecP, unsigned int* newP) {

	//time1 = clock();
	unsigned int* C;
	unsigned int* P;
	//numI = 1;
	//printf("numI: %u\n", numI);

	// have each block 
	if(threadIdx.x == 0){
		//printf("const n: %u\n", constN[1]);
		nodes = 0;
		maxSize = 0;
		max[blockIdx.x] = 0;	
		C = devC + (numI * blockIdx.x);
		P = devP + (numI * blockIdx.x);
	}
	//printf("addr C: %p %d\n", C, blockIdx.x);
	//printf("addr P: %p %d\n", P, blockIdx.x);

	/*if(threadIdx.x == 0 && blockIdx.x == 0){
		for(int i=0; i<gridDim.x; i++){
			printf("Block: %d C: \n", blockIdx.x);
			printBitSet(C, numI);
			printf("\n");
			printf("Block: %d P: \n", blockIdx.x);
			printBitSet(P, numI);
			printf("\n");
			C += numI;
			P += numI;	
		}
	}*/
	if(threadIdx.x == 0){
		copyBitSet(devRecC + (blockIdx.x * numV * numI), C);
		copyBitSet(devRecP + (blockIdx.x * numV * numI), P);
		copyBitSet(newP + (blockIdx.x * numV * numI), P);
		recSearchP(N, solution, max, devRecC + (blockIdx.x * numV * numI) + numI, devRecP + (blockIdx.x * numV * numI) + numI, newP + (blockIdx.x * numV * numI));
	}

	//if(blockIdx.x == 0){
		//printf("Max found: %d\n", maxSize);
	//}
}

__device__ void recSearchP(unsigned int* N,
 unsigned int* solution, unsigned int* max, unsigned int* C, unsigned int* P, unsigned int* newP){

	// copy C and P
	if(threadIdx.x == 0){
		copyBitSet(C, C-numI);
		copyBitSet(P, newP);
		newP += numI;
		nodes++;

		int m = getSetBitCount(P);
		unsigned int U[200];
		unsigned int color[200];

		colorVertsP(P, U, color);

		// iterate over the candidate set
		for(int i=m-1; i>=0; i--){

			if(color[i] + getSetBitCount(C) <= maxSize){
				return;
			}

			// copy the candidate set
			copyBitSet(newP, P);
			int v = U[i];
			setBit(C[v/32], v%32);

			intersectBitSet(newP, &N[v * numI]);

		/*printf("new P: \n");
		printBitSet(newP, numI);
		printf("\n");*/

			// if maximal, check for maximum
			if(getSetBitCount(newP) == 0 && getSetBitCount(C) > maxSize){
				// save the solution
				atomicMax(&maxSize, getSetBitCount(C));
				//printf("b: %d, m: %d\n", blockIdx.x, getSetBitCount(C));
				max[blockIdx.x] = getSetBitCount(C);
				//printf("size: %d\n", maxSize);
				copyBitSet(solution + (blockIdx.x*numI), C);
				/*printf("Solution: ");
				printBitSet(solution + (blockIdx.x*numI), numI);
				printf("\n");*/
			}
			else if(getSetBitCount(newP) > 0){
				recSearchP(N, solution, max, C + numI, P + numI, newP);
			}

			// remove v from P and C
			clearBit(C[v/32], v%32);
			clearBit(P[v/32], v%32);

		}

		//delete[] U;
		//delete[] color;
	}
	/*if(C == NULL || P == NULL){
		printf("Out of heap memory\n");
		return;
	}
	if(oldP == NULL || oldC == NULL){
		printf("Out of stack memory\n");
		return;
	}	*/

	//printf("addr P: %p\n", oldP);

	/*if(nodes > 2)
		return;*/

	/*printf("cuda m: %d\n", m);
	printf("P: \n");
	printBitSet(P, numI);
	printf("\n");*/

	/*printf("cuda U:\n");
	for(int i=0; i<m; i++){
		printf("%u ", U[i]);
	}
	printf("\n");
	printf("cuda color:\n");
	for(int i=0; i<m; i++){
		printf("%u ", color[i]);
	}
	printf("\n");*/
}

__device__ void colorVertsP(unsigned int* P, unsigned int* U, unsigned int* color){
	// copy the candidate set
	//printf("thread: %d\n", threadIdx.x);
	if(threadIdx.x == 0){
		unsigned int copyP[32];
		unsigned int Q[32];
		copyBitSet(copyP, P);
		int v;
		int colorClass = 0;
		int i = 0;

		while(getSetBitCount(copyP) != 0){
			colorClass++;

			// copy the candidate set
			copyBitSet(Q, copyP);

			while(getSetBitCount(Q) != 0){
				// return the index of the first set bit
				v = findFirstBit(Q);
				//printf("%d - %d\n", colorClass, v);
				// remove v from Q and copyP
				clearBit(copyP[v/32], v%32);
				clearBit(Q[v/32], v%32);

				/*if(v == 43 && colorClass == 2){
					printf("Q: ");
					printBitSet(Q, numI);
					printf("\n");
				}*/

				intersectBitSet(Q, &constInvN[v * numI]);

				/*if(v == 43 && colorClass == 2){
					printf("Q2: ");
					printBitSet(Q, numI);
					printf("\n");
					printf("first bit: %d\n", __ffs(Q[2]));
				}*/

				U[i] = v;
				color[i++] = colorClass;
			}
		}
	}
}

__device__ void copyBitSetP(unsigned int* dest, unsigned int* src){
	dest[threadIdx.x] = src[threadIdx.x];
}