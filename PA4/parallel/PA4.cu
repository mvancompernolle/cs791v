#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cstdlib>
#include <sys/time.h>
#include <time.h>

#include "reduce.h"

// used to take the average run time on the gpu
#define NUM_ITERATIONS 3

enum RUN_TYPE {CPU, RECURSIVE_HOST, RECURSIVE_DEVICE};

struct Range{
  unsigned int start;
  unsigned int end;

  Range(){
    start = 2;
    end = 2;
  }

  void printRange(){
    if(end > start)
      std::cout << start << " - " << end;
    else
      std::cout << start;
  }
};

void incrByHalfPowerOf2(int& num){
  if(num != 1)
    num += pow(2, ceil(log(num+1)/log(2)))/4;
  else
    num++;
}

long long int todiff(struct timeval *tod1, struct timeval *tod2)
{
  long long t1, t2;
  t1 = tod1->tv_sec * 1000000 + tod1->tv_usec;
  t2 = tod2->tv_sec * 1000000 + tod2->tv_usec;
  return t1 - t2;
}


void getCudaInfo(hipDeviceProp_t props[], int& count){

  hipError_t err;
  err = hipGetDeviceCount(&count);
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }

  for(int i=0; i<count; i++){
    err = hipGetDeviceProperties(&props[i], i);
    if (err != hipSuccess) {
      std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
      exit(1);
    }
  }
}

void runTest(Range numElements, Range numThreads, Range numBlocks, RUN_TYPE reductionType){
  float elapsedTime, totalTime;
  struct timeval tod1, tod2;
  hipEvent_t start, end;
  std::ofstream fout;
  int correctSum = 0;
  int *input, *partialSums;
  int *devInput, *devPartialSums;
  int dev_result = 0;
  int currentSize;


  fout.open("results.txt", std::ios::app);

  hipEventCreate(&start);
  hipEventCreate(&end);

  // loop for the number of elements
  for(int n = numElements.start; n <= numElements.end; incrByHalfPowerOf2(n)){

    fout << std::endl << "Size: " << n << " Type: ";
    switch(reductionType){
      case CPU:
        fout << "CPU Reduction";
        break;
      case RECURSIVE_HOST:
        fout << "Recursive Host Reduction";
        break;
      case RECURSIVE_DEVICE:
        fout << "Recursive Device Reduction";
        break;
    }
    fout << std::endl;

    for(int t = numThreads.start; t <= numThreads.end; incrByHalfPowerOf2(t)){
      fout << "," << t;
    }
    fout << std::endl;

    // Arrays on the host (CPU)
    input = new int[n];

    // arrays on device (GPU)
    hipError_t err = hipMalloc( (void**) &devInput, n * sizeof(int));
    if (err != hipSuccess) {
      std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
      exit(1);
    }

    correctSum = 0;
    for (int i = 0; i < n; ++i) {
      input[i] = rand() % 1 + 1;
      correctSum += input[i];
    }
    //std::cout << "Correct Reduced Sum: " << correctSum << std::endl;

    // loop for the number of blocks
    for(int b = numBlocks.start; b <= numBlocks.end; incrByHalfPowerOf2(b)){

      if(b == 65536)
        b = 65535;

      fout << b << ", ";

      partialSums = new int[b];

      err = hipMalloc( (void**) &devPartialSums, (b) * sizeof(int));
      if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
      }

      // loop for the number of threads
      for(int t = numThreads.start; t <= numThreads.end; incrByHalfPowerOf2(t)){

        totalTime = 0;

        // loop for number of iterations
        for(int i = 0; i < NUM_ITERATIONS; i++){
          hipEventRecord( start, 0 );

          err = hipMemcpy(devInput, input, n * sizeof(int), hipMemcpyHostToDevice);
          if (err != hipSuccess) {
            std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
            exit(1);
          }

          currentSize = n;
          dev_result = 0;

          int memorySize = (t <= 32) ? 2 * t * sizeof(int) : t * sizeof(int);

          // run normal reduction for CPU final reduction versions
          if(reductionType == CPU || reductionType == RECURSIVE_HOST)
            reduce<<<b, t, memorySize>>>(devInput, devPartialSums, n, t);
          // call the recursive kernel for device version
          else if(reductionType == RECURSIVE_DEVICE)
          {
            reduceRecursive<<<b, t, memorySize>>>(devInput, devPartialSums, n, t);
          }

          if(reductionType == RECURSIVE_HOST){
            // call reduce until completely reduced
            currentSize = b;
            while(currentSize > 1){
              reduce<<<currentSize, t, memorySize>>>(devPartialSums, devPartialSums, currentSize, t);
              currentSize = (currentSize + (t*2 - 1))/(t*2);
              //std::cout << "currSize: " << currentSize << std::endl;
            }
          }

          // get result back
          hipMemcpy(partialSums, devPartialSums, b * sizeof(int), hipMemcpyDeviceToHost);

          hipEventRecord( end, 0 );
          hipEventSynchronize( end );
          hipEventElapsedTime( &elapsedTime, start, end );
          totalTime += elapsedTime;

          // get time to reduce on CPU
          if(reductionType == CPU){
            gettimeofday(&tod1, NULL);
            for(int i = 0; i < b; i++){
              dev_result += partialSums[i];
            }
            gettimeofday(&tod2, NULL);
            totalTime += todiff(&tod2, &tod1)/1000;
          }
          // perform final reduction for recursive host
          else if(reductionType == RECURSIVE_HOST){
            for(int index=0; index<currentSize; index++){
              dev_result += partialSums[index];
            }
          }
          // perfor final reduction for recursive device
          else{
            /*for(int index=0; index<b; index++){
              std::cout << "num " << partialSums[index] << std::endl;
            }*/
            dev_result = partialSums[0];
          }


          //std::cout << "blocks: " << b << " t: " << t << " n: " << n << std::endl;
          //std::cout << "Device sum: " << dev_result << std::endl;

          if(dev_result != correctSum){
            std::cout << "Results did not match! " << dev_result << std::endl;

            // clean up events - we should check for error codes here.
            // hipEventDestroy( start );
            // hipEventDestroy( end );

            // // clean up device pointers
            // hipFree(devInput);
            // hipFree(devPartialSums);
            //exit(1);
          }
        } // end of iterations loop

        // print results to screen
        std::cout << "Size: " << n << " Blocks: " << b << " Threads: " << t << std::endl;
        //std::cout << totalCPUTime/1000 << " " << totalTime << std::endl;
        std::cout << "Your program took: " << totalTime/NUM_ITERATIONS << " ms (I/O). " << std::endl << std::endl;

        // output results to file
        //fout << n << ", " << b << ", " << t << ", " << totalTime/NUM_ITERATIONS << std::endl;
        fout << totalTime/NUM_ITERATIONS << ",";

      } // end of threads loop

      fout << std::endl;

      delete []partialSums;
      hipFree(devPartialSums);

    } // end blocks loop

    delete []input;
    hipFree(devInput);

  } // end of vector size loop

  // Cleanup in the event of success.
  hipEventDestroy(start);
  hipEventDestroy(end);

  fout.close();
}


int main(int argc, char *argv[]) {

  hipDeviceProp_t props[2];
  int count = 1;
  char input;
  bool running = true; 
  Range numThreads, numElements, numBlocks;
  unsigned int currentDevice = 0;
  char inputc;
  int inputi;
  RUN_TYPE reductionType = CPU;
  numThreads.start = 1024;
  numThreads.end = 1024;
  numElements.start = 10000;
  numElements.end = 10000;
  numBlocks.start = 1024;
  numBlocks.end = 1024;

  srand(time(NULL));

  // get cuda info
  getCudaInfo(props, count);

  // if in sli, set cuda to use device 2 (more memory available)
  if(count == 2){
    currentDevice = 1;
    hipSetDevice(currentDevice);
  }

  do{
    // print menu options
    std::cout << std::endl << "-------------- VECTOR REDUCTION MENU ---------------" << std::endl;
    std::cout << "1. Select the vector input size" << std::endl;
    std::cout << "2. Select the number of threads" << std::endl;
    std::cout << "3. Select the number of blocks" << std::endl;
    std::cout << "4. Select Reduction type" << std::endl;
    std::cout << "5. Display run settings" << std::endl;
    std::cout << "6. Run vector reduction" << std::endl;
    std::cout << "Q. Quit Program" << std::endl;
    std::cout << "Select a menu option: ";
    std::cin >> input;
    std::cout << std::endl;

    switch(input){
      case '1':
        std::cout << "Select an input size: ";
        std::cin >> numElements.start;
        std::cout << "Do you want the input size to loop by half powers of two? (Y/N): ";
        std::cin >> inputc;
        if(inputc == 'y' || inputc == 'Y'){
          std::cout << "Select an input size to stop looping at: ";
          std::cin >> inputi;
          numElements.end = inputi;
        }
        else if(inputc == 'n' || inputc == 'N'){
          numElements.end = numElements.start;
        }
        break;

      case '2':
        std::cout << "Select the number of threads: ";
        std::cin >> numThreads.start;
        std::cout << "Do you want the the number of threads to loop by half powers of two? (Y/N): ";
        std::cin >> inputc;
        if(inputc == 'y' || inputc == 'Y'){
          std::cout << "Select a number of threads to stop looping at: ";
          std::cin >> inputi;
          numThreads.end = inputi;
        }
        else if(inputc == 'n' || inputc == 'N'){
          numThreads.end = numThreads.start;
        }
        break;

      case '3':
        std::cout << "Select the number of blocks: ";
        std::cin >> numBlocks.start;
        std::cout << "Do you want the the number of blocks to loop by half powers of two? (Y/N): ";
        std::cin >> inputc;
        if(inputc == 'y' || inputc == 'Y'){
          std::cout << "Select a number of blocks to stop looping at: ";
          std::cin >> inputi;
          numBlocks.end = inputi;
        }
        else if(inputc == 'n' || inputc == 'N'){
          numBlocks.end = numBlocks.start;
        }
        break;

      case '4':
        std::cout << "Select the type of reduction you want (1. CPU | 2. Recursive Host | 3. Recursive Device)" << std::endl;
        std::cin >> inputi;
        switch(inputi){
          case 1:
            reductionType = CPU;
            break;
          case 2:
            reductionType = RECURSIVE_HOST;
            break;
          case 3:
            reductionType = RECURSIVE_DEVICE;
            break;
          default:
            reductionType = CPU;
        }
        break;

      case '5':
        std::cout << "------------- RUN INFO --------------" << std::endl;
        std::cout << "Vector Size:\t\t"; 
        numElements.printRange();
        std::cout << std::endl;
        std::cout << "Number of threads:\t"; 
        numThreads.printRange();
        std::cout << std::endl;
        std::cout << "Number of blocks:\t"; 
        numBlocks.printRange();
        std::cout << std::endl;
        std::cout << "Reduction Type:\t\t"; 
        switch(reductionType){
          case CPU:
            std::cout << "CPU Reduction";
            break;
          case RECURSIVE_HOST:
            std::cout << "Recursive Host Reduction";
            break;
          case RECURSIVE_DEVICE:
            std::cout << "Recursive Device Reduction";
            break;
        }
        std::cout << std::endl;
        break;
      case '6':
        runTest(numElements, numThreads, numBlocks, reductionType);
        break;
      case 'Q': 
      case 'q':
        running = false;
        break;
    }

  }while(running);
}
