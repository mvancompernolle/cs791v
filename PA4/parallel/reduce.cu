#include "hip/hip_runtime.h"

#include "reduce.h"

// REFERENCE : http://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

__device__ void warpWiseReduce(volatile int* sdata, const unsigned int tid, const unsigned int blockSize){
  // reduce without sync (threads in warp all execute at same time)
  if(blockSize >= 64)
    sdata[tid] += sdata[tid + 32];

  if(blockSize >= 32)
    sdata[tid] += sdata[tid + 16];

  if(blockSize >= 16)
    sdata[tid] += sdata[tid + 8];

  if(blockSize >= 8)
    sdata[tid] += sdata[tid + 4];

  if(blockSize >= 4)
    sdata[tid] += sdata[tid + 2];

  if(blockSize >= 2)
    sdata[tid] += sdata[tid + 1];
}

__global__ void reduce(const int *input, int *blockResults, const unsigned int N, const unsigned int blockSize) {

  extern __shared__ int sharedData[];
  int i = blockIdx.x*(blockSize*2) + threadIdx.x;
  int gridStep = blockSize*2*gridDim.x;
  sharedData[threadIdx.x] = 0;

  int sum = 0;

  // attempt to perform an initial reduce on loading for performance
  // stride so that an input vector larger than the total shared memory between blocks still fits
  while(i < N){

    sum += input[i];

    if(i + blockSize < N)
      sum += input[i + blockSize];

    i += gridStep;
  }
  sharedData[threadIdx.x] = sum;

  // sync to make sure all shared memory has been initialized
  __syncthreads();

  // repeatedly sum the left half of shared memory with the right half
  // effectively reduces by half of current size each time and leaves half of threads idle
  // make sure to sync between reducitons
  if(blockSize >= 1024 && threadIdx.x < (blockSize + (blockSize/2 -1))/2){
    sharedData[threadIdx.x] += sharedData[threadIdx.x + (blockSize + (blockSize/2 -1))/2];
  }
  __syncthreads();

  if(blockSize >= 512 && threadIdx.x < (blockSize + (blockSize/4 -1))/4){
    sharedData[threadIdx.x] += sharedData[threadIdx.x + (blockSize + (blockSize/4 -1))/4];
  }
  __syncthreads();

  if(blockSize >= 256 && threadIdx.x < (blockSize + (blockSize/8 -1))/8){
    sharedData[threadIdx.x] += sharedData[threadIdx.x + (blockSize + (blockSize/8 -1))/8];
  }
  __syncthreads();

  if(blockSize >= 128 && threadIdx.x < (blockSize + (blockSize/16 -1))/16){
    sharedData[threadIdx.x] += sharedData[threadIdx.x + (blockSize + (blockSize/16 -1))/16];
  }
  __syncthreads();

  // when small enough for a warp to reduce
  if(threadIdx.x < 32){
    warpWiseReduce(sharedData, threadIdx.x, blockSize);
  }

  // save block's partial sum in results
  if(threadIdx.x == 0){
    blockResults[blockIdx.x] = sharedData[0];
  }
}

__global__ void reduceRecursive(const int *input, int *blockResults, const unsigned int N, const unsigned int blockSize) {

  extern __shared__ int sharedData[];
  int i = blockIdx.x*(blockSize*2) + threadIdx.x;
  int gridStep = blockSize*2*gridDim.x;
  sharedData[threadIdx.x] = 0;

  int sum = 0;

  // attempt to perform an initial reduce on loading for performance
  // stride so that an input vector larger than the total shared memory between blocks still fits
  while(i < N){

    sum += input[i];

    if(i + blockSize < N)
      sum += input[i + blockSize];

    i += gridStep;
  }
  sharedData[threadIdx.x] = sum;

  // sync to make sure all shared memory has been initialized
  __syncthreads();

  // repeatedly sum the left half of shared memory with the right half
  // effectively reduces by half of current size each time and leaves half of threads idle
  // make sure to sync between reducitons
  if(blockSize >= 1024 && threadIdx.x < 512){
    sharedData[threadIdx.x] += sharedData[threadIdx.x + 512];
  }
  __syncthreads();

  if(blockSize >= 512 && threadIdx.x < 256){
    sharedData[threadIdx.x] += sharedData[threadIdx.x + 256];
  }
  __syncthreads();

  if(blockSize >= 256 && threadIdx.x < 128){
    sharedData[threadIdx.x] += sharedData[threadIdx.x + 128];
  }
  __syncthreads();

  if(blockSize >= 128 && threadIdx.x < 64){
    sharedData[threadIdx.x] += sharedData[threadIdx.x + 64];
  }
  __syncthreads();

  // when small enough for a warp to reduce
  if(threadIdx.x < 32){
    warpWiseReduce(sharedData, threadIdx.x, blockSize);
  }

  // save block's partial sum in results
  if(threadIdx.x == 0){
    blockResults[blockIdx.x] = sharedData[0];

    if(blockIdx.x == 0 && N > 1){
      int b = (gridDim.x + (blockSize*2 - 1))/(blockSize*2);
      int memSize = (blockSize <= 32) ? 2 * blockSize * sizeof(int) : blockSize * sizeof(int);
      reduceRecursive<<<b, blockSize, memSize>>>(blockResults, blockResults, b, blockSize);
      hipDeviceSynchronize();
    }

  }
}

