#include "hip/hip_runtime.h"

#include "reduce.h"

// REFERENCE : http://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

__global__ void reduce(const int *input, int *blockResults, const unsigned int N, const unsigned int blockSize) {

  extern __shared__ int sharedData[];
  int i = blockIdx.x*(blockSize*2) + threadIdx.x;
  int gridStep = blockSize*2*gridDim.x;
  sharedData[threadIdx.x] = 0;

  int sum = 0;

  // attempt to perform an initial reduce on loading for performance
  // stride so that an input vector larger than the total shared memory between blocks still fits
  while(i < N){

    sum += input[i];

    if(i + blockSize < N)
      sum += input[i + blockSize];

    i += gridStep;
  }
  sharedData[threadIdx.x] = sum;

  // sync to make sure all shared memory has been initialized
  __syncthreads();

  // loop version
  int size = blockSize;
  for(unsigned int s=blockDim.x/2; s>0; s/=2){
    __syncthreads();

    // add up left half with right half of current reduction
    if(threadIdx.x < s){
      sharedData[threadIdx.x] += sharedData[threadIdx.x+s];

      // have the first thread do one additional add if the current size is odd
      if(size&0x0001 == 0x0001 && threadIdx.x == 0)
        sharedData[threadIdx.x] += sharedData[size-1];
    }

    size /= 2;
  }

  __syncthreads();

  // save block's partial sum in results
  if(threadIdx.x == 0){
    blockResults[blockIdx.x] = sharedData[0];
  }
}

__device__ unsigned int count = 0;
__shared__ bool isLastBlockDone;
__global__ void reduceRecursive(const int *input, int *blockResults, const unsigned int N, const unsigned int blockSize) {

  extern __shared__ int sharedData[];
  int i = blockIdx.x*(blockSize*2) + threadIdx.x;
  int gridStep = blockSize*2*gridDim.x;
  sharedData[threadIdx.x] = 0;

  int sum = 0;

  // attempt to perform an initial reduce on loading for performance
  // stride so that an input vector larger than the total shared memory between blocks still fits
  while(i < N){

    sum += input[i];

    if(i + blockSize < N)
      sum += input[i + blockSize];

    i += gridStep;
  }
  sharedData[threadIdx.x] = sum;

  // sync to make sure all shared memory has been initialized
  __syncthreads();

  // loop version
  int size = blockSize;
  for(unsigned int s=blockDim.x/2; s>0; s/=2){
    __syncthreads();

    // add up left half with right half of current reduction
    if(threadIdx.x < s){
      sharedData[threadIdx.x] += sharedData[threadIdx.x+s];

      // have the first thread do one additional add if the current size is odd
      if(size&0x0001 == 0x0001 && threadIdx.x == 0)
        sharedData[threadIdx.x] += sharedData[size-1];
    }

    size /= 2;
  }

  __syncthreads();

  // save block's partial sum in results
  if(threadIdx.x == 0){
    blockResults[blockIdx.x] = sharedData[0];

    __threadfence();

    unsigned int value = atomicInc(&count, gridDim.x);
    isLastBlockDone = (value == (gridDim.x-1));
  }

  // make sure each block has correct isLastBlockDone value
  __syncthreads();

  // recursively call once all of the blocks have finished
  if(isLastBlockDone && threadIdx.x == 0 && N > 1){

    // reset flag and count
    isLastBlockDone = false;
    count = 0;

    // recalculate the input size and amount of memeory needed
    int b = (gridDim.x + (blockSize*2 - 1))/(blockSize*2);
    int memSize = (blockSize <= 32) ? 2 * blockSize * sizeof(int) : blockSize * sizeof(int);

    // call reduce again recursively
    reduceRecursive<<<b, blockSize, memSize>>>(blockResults, blockResults, gridDim.x, blockSize);
  }

}

           